#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
* Copyright (c) 2013, Georgia Institute of Technology
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice, this
* list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright notice,
* this list of conditions and the following disclaimer in the documentation
* and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/**********************************************
 * @file mppi_controller.cu
 * @author Grady Williams <gradyrw@gmail.com>
 * @date May 24, 2017
 * @copyright 2017 Georgia Institute of Technology
 * @brief Implementation of the mppi_controller class.
 ***********************************************/

/******************************************************************************
//MPPI Kernel Implementations and helper launch files
*******************************************************************************/

#define BLOCKSIZE_X MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::BLOCKSIZE_X
#define BLOCKSIZE_Y MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::BLOCKSIZE_Y
#define BLOCKSIZE_WRX MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::BLOCKSIZE_WRX
#define STATE_DIM DYNAMICS_T::STATE_DIM
#define CONTROL_DIM DYNAMICS_T::CONTROL_DIM
#define SHARED_MEM_REQUEST_GRD DYNAMICS_T::SHARED_MEM_REQUEST_GRD
#define SHARED_MEM_REQUEST_BLK DYNAMICS_T::SHARED_MEM_REQUEST_BLK
#define NUM_ROLLOUTS MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::NUM_ROLLOUTS

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
__global__ void rolloutKernel(int num_timesteps, float* state_d, float* U_d, float* du_d, float* nu_d, 
                              float* costs_d, DYNAMICS_T dynamics_model, COSTS_T mppi_costs, 
                              int opt_delay)
{
  int i,j;
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  int bdx = blockIdx.x;

  //Initialize the local state, controls, and noise
  float* s;
  float* s_der;
  float* u;
  float* nu;
  float* du;
  int* crash;

  //Create shared arrays for holding state and control data.
  __shared__ float state_shared[BLOCKSIZE_X*STATE_DIM];
  __shared__ float state_der_shared[BLOCKSIZE_X*STATE_DIM];
  __shared__ float control_shared[BLOCKSIZE_X*CONTROL_DIM];
  __shared__ float control_var_shared[BLOCKSIZE_X*CONTROL_DIM];
  __shared__ float exploration_variance[BLOCKSIZE_X*CONTROL_DIM];
  __shared__ int crash_status[BLOCKSIZE_X];
  //Create a shared array for the dynamics model to use
  __shared__ float theta[SHARED_MEM_REQUEST_GRD + SHARED_MEM_REQUEST_BLK*BLOCKSIZE_X];

  //Initialize trajectory cost
  float running_cost = 0;

  //Initialize the dynamics model.
  dynamics_model.cudaInit(theta);

  int global_idx = BLOCKSIZE_X*bdx + tdx;
  if (global_idx < NUM_ROLLOUTS) {
    //Portion of the shared array belonging to each x-thread index.
    s = &state_shared[tdx*STATE_DIM];
    s_der = &state_der_shared[tdx*STATE_DIM];
    u = &control_shared[tdx*CONTROL_DIM];
    du = &control_var_shared[tdx*CONTROL_DIM];
    nu = &exploration_variance[tdx*CONTROL_DIM];
    crash = &crash_status[tdx];
    //Load the initial state, nu, and zero the noise
    for (i = tdy; i < STATE_DIM; i+= blockDim.y) {
      s[i] = state_d[i];
      s_der[i] = 0;
    }
    //Load nu
    for (i = tdy; i < CONTROL_DIM; i+= blockDim.y) {
      u[i] = 0;
      du[i] = 0;
      nu[i] = nu_d[i];
    }
    crash[0] = 0;
  }
  __syncthreads();
  /*<----Start of simulation loop-----> */
  for (i = 0; i < num_timesteps; i++) {
    if (global_idx < NUM_ROLLOUTS) {
      for (j = tdy; j < CONTROL_DIM; j+= blockDim.y) {
        //Noise free rollout
        if (global_idx == 0 || i < opt_delay) { //Don't optimize variables that are already being executed
          du[j] = 0.0;
          u[j] = U_d[i*CONTROL_DIM + j];
        }
        else if (global_idx >= .99*NUM_ROLLOUTS) {
          du[j] = du_d[CONTROL_DIM*num_timesteps*(BLOCKSIZE_X*bdx + tdx) + i*CONTROL_DIM + j]*nu[j];
          u[j] = du[j];
        }
        else {
          du[j] = du_d[CONTROL_DIM*num_timesteps*(BLOCKSIZE_X*bdx + tdx) + i*CONTROL_DIM + j]*nu[j];
          u[j] = U_d[i*CONTROL_DIM + j] + du[j];
        }
        du_d[CONTROL_DIM*num_timesteps*(BLOCKSIZE_X*bdx + tdx) + i*CONTROL_DIM + j] = u[j];
      }
    }
    __syncthreads();
    if (tdy == 0 && global_idx < NUM_ROLLOUTS){
       dynamics_model.enforceConstraints(s, u);
    }
    __syncthreads();
    //Compute the cost of the being in the current state
    if (tdy == 0 && global_idx < NUM_ROLLOUTS && i > 0 && crash[0] > -1) {
      //Running average formula
      running_cost += (mppi_costs.computeCost(s, u, du, nu, crash, i) - running_cost)/(1.0*i);
    }
    //Compute the dynamics
    if (global_idx < NUM_ROLLOUTS){
      dynamics_model.computeStateDeriv(s, u, s_der, theta);
    }
    __syncthreads();
    //Update the state
    if (global_idx < NUM_ROLLOUTS){
      dynamics_model.incrementState(s, s_der);
    }
    //Check to see if the rollout will result in a (physical) crash.
    if (tdy == 0 && global_idx < NUM_ROLLOUTS) {
      mppi_costs.getCrash(s, crash);
    }
  }
  /* <------- End of the simulation loop ----------> */
  if (global_idx < NUM_ROLLOUTS && tdy == 0) {   //Write cost results back to global memory.
    costs_d[(BLOCKSIZE_X)*bdx + tdx] = running_cost + mppi_costs.terminalCost(s);
  }
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
__global__ void normExpKernel(float* state_costs_d, float gamma, float baseline)
{
  int tdx = threadIdx.x;
  int bdx = blockIdx.x;
  if (BLOCKSIZE_X*bdx + tdx < NUM_ROLLOUTS) {
    float cost2go = 0;
    cost2go = state_costs_d[BLOCKSIZE_X*bdx + tdx] - baseline;
    state_costs_d[BLOCKSIZE_X*bdx + tdx] = exp(-gamma*cost2go);
  }
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
__global__ void weightedReductionKernel(float* states_d, float* du_d, float* nu_d, 
                                        float normalizer, int num_timesteps)
{
  int tdx = threadIdx.x;
  int bdx = blockIdx.x;

  __shared__ float u_system[STATE_DIM*((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1)];
  int stride = BLOCKSIZE_WRX;

  float u[CONTROL_DIM];

  int i,j;
  for (i = 0; i < CONTROL_DIM; i++) {
    u[i] = 0;
  }

  for (j = 0; j < CONTROL_DIM; j++) {
    u_system[tdx*CONTROL_DIM + j] = 0;
  }
  __syncthreads();

  if (BLOCKSIZE_WRX*tdx < NUM_ROLLOUTS) {
    float weight = 0;
    for (i = 0; i < stride; i++) {
      if (stride*tdx + i < NUM_ROLLOUTS) {
        weight = states_d[stride*tdx + i]/normalizer;
        for (j = 0; j < CONTROL_DIM; j++) {
          u[j] = du_d[(stride*tdx + i)*(num_timesteps*CONTROL_DIM) + bdx*CONTROL_DIM + j];
          u_system[tdx*CONTROL_DIM + j] += weight*u[j];
        }
      }
    }
  }
  __syncthreads();
  if (tdx == 0 && bdx < num_timesteps) {
    for (i = 0; i < CONTROL_DIM; i++) {
      u[i] = 0;
    }
    for (i = 0; i < (NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1; i++) {
      for (j = 0; j < CONTROL_DIM; j++) {
        u[j] += u_system[CONTROL_DIM*i + j];
      }
    }
    for (i = 0; i < CONTROL_DIM; i++) {
      du_d[CONTROL_DIM*bdx + i] = u[i];
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void launchRolloutKernel(int num_timesteps, float* state_d, float* U_d, float* du_d, float* nu_d, 
                         float* costs_d, DYNAMICS_T *dynamics_model, COSTS_T *mppi_costs, 
                         int opt_delay, hipStream_t stream)
{
  const int GRIDSIZE_X = (NUM_ROLLOUTS-1)/BLOCKSIZE_X + 1;
  //transferMemToConst(dynamics_model.theta_d_);
  dim3 dimBlock(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
  dim3 dimGrid(GRIDSIZE_X, 1, 1);
  //printf("%d \n", sizeof(DYNAMICS_T));
  //int dev;
  //hipGetDevice(&dev);
  //printf("Device: %d \n", dev);
  //HANDLE_ERROR(hipMemPrefetchAsync(dynamics_model, sizeof(DYNAMICS_T), dev, stream) );
  //HANDLE_ERROR(hipMemPrefetchAsync(dynamics_model->control_rngs_d_, 2*sizeof(float2), dev, stream) );
  rolloutKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y><<<dimGrid, dimBlock, 0, stream>>>(num_timesteps, state_d, U_d, 
    du_d, nu_d, costs_d, *dynamics_model, *mppi_costs, opt_delay);
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void launchNormExpKernel(float* costs_d, float gamma, float baseline, hipStream_t stream)
{
  dim3 dimBlock(BLOCKSIZE_X, 1, 1);
  dim3 dimGrid((NUM_ROLLOUTS-1)/BLOCKSIZE_X + 1, 1, 1);
  normExpKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y><<<dimGrid, dimBlock, 0, stream>>>(costs_d, gamma, baseline);
}

//Launches the multiplication and reduction kernel
template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void launchWeightedReductionKernel(float* state_costs_d, float* du_d, float* nu_d, 
                                  float normalizer, int num_timesteps, hipStream_t stream)
{
    dim3 dimBlock((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1, 1, 1);
    dim3 dimGrid(num_timesteps, 1, 1);
    weightedReductionKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y><<<dimGrid, dimBlock, 0, stream>>>(state_costs_d, du_d, nu_d, normalizer, num_timesteps);
}

#undef BLOCKSIZE_X
#undef BLOCKSIZE_Y
#undef BLOCKSIZE_WRX
#undef STATE_DIM
#undef CONTROL_DIM
#undef SHARED_MEM_REQUEST_GRD
#undef SHARED_MEM_REQUEST_BLK
#undef NUM_ROLLOUTS


/******************************************************************************************************************
MPPI Controller implementation
*******************************************************************************************************************/

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::MPPIController(DYNAMICS_T* model, COSTS_T* costs, 
                                                                              int num_timesteps, int hz, float gamma, 
                                                                              float* exploration_var, float* init_u, 
                                                                              int num_optimization_iters, int opt_stride,
                                                                              hipStream_t stream)
{
  //Initialize internal classes which use the CUDA API.
  model_ = model;
  costs_ = costs;
  hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen_, 1234ULL);

  //Set the CUDA stream, and attach unified memory to the particular stream.
  //This must be done AFTER all internal classes that use unified memory are initialized (cost and model)
  setCudaStream(stream);

  //Initialize parameters, including the number of rollouts and timesteps
  hz_ = hz;
  numTimesteps_ = num_timesteps;
  optimizationStride_ = opt_stride;
  gamma_ = gamma;
  num_iters_ = num_optimization_iters;

  //Initialize host vectors
  nu_.assign(exploration_var, exploration_var + CONTROL_DIM);
  init_u_.assign(init_u, init_u + CONTROL_DIM);
  control_hist_.assign(2*CONTROL_DIM, 0);
  state_solution_.assign(numTimesteps_*STATE_DIM, 0);
  control_solution_.assign(numTimesteps_*CONTROL_DIM, 0);
  du_.resize(numTimesteps_*CONTROL_DIM);
  U_.resize(numTimesteps_*CONTROL_DIM);
  traj_costs_.resize(NUM_ROLLOUTS);

  //Allocate memory on the device.
  allocateCudaMem();
  //Transfer exploration variance to device.
  HANDLE_ERROR(hipMemcpyAsync(nu_d_, nu_.data(), CONTROL_DIM*sizeof(float), hipMemcpyHostToDevice, stream_));
  //Get the parameters for the control input and initialize the sequence.
  initDDP();
  resetControls();
  //Make sure all cuda operations have finished.
  hipStreamSynchronize(stream_);
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::~MPPIController()
{}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::setCudaStream(hipStream_t stream)
{
  //Set the CUDA stream and attach unified memory in object to that stream
  stream_ = stream;
  model_->bindToStream(stream_);
  costs_->bindToStream(stream_);
  hiprandSetStream(gen_, stream_);
} 

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::allocateCudaMem()
{
  HANDLE_ERROR( hipMalloc((void**)&state_d_, STATE_DIM*sizeof(float)));
  HANDLE_ERROR( hipMalloc((void**)&nu_d_, STATE_DIM*sizeof(float)));
  HANDLE_ERROR( hipMalloc((void**)&traj_costs_d_, NUM_ROLLOUTS*sizeof(float)));
  HANDLE_ERROR( hipMalloc((void**)&U_d_, CONTROL_DIM*numTimesteps_*sizeof(float)));
  HANDLE_ERROR( hipMalloc((void**)&du_d_, NUM_ROLLOUTS*numTimesteps_*CONTROL_DIM*sizeof(float)));
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::deallocateCudaMem(){
  hipFree(state_d_);
  hipFree(nu_d_);
  hipFree(traj_costs_d_);
  hipFree(U_d_);
  hipFree(du_d_);
  //Free cuda memory used by the model and costs.
  model_->freeCudaMem();
  costs_->freeCudaMem();
  hipStreamDestroy(stream_);
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::initDDP()
{
  util::DefaultLogger logger;
  bool verbose = false;
  ddp_model_ = new ModelWrapperDDP<DYNAMICS_T>(model_);
  ddp_solver_ = new DDP<ModelWrapperDDP<DYNAMICS_T>>(1.0/hz_, numTimesteps_, 1, &logger, verbose);

  Q_.setIdentity();
  Q_.diagonal() << 0.5, 0.5, 0.25, 0.0, 0.05, 0.01, 0.01;

  Qf_.setIdentity();
  Qf_.diagonal() << 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0;

  R_.setIdentity();
  R_.diagonal() << 10.0, 10.0;

  U_MIN_ << model_->control_rngs_[0].x, model_->control_rngs_[1].x;
  U_MAX_ << model_->control_rngs_[0].y, model_->control_rngs_[1].y;
  
  //Define the running and terminal cost
  run_cost_ = new TrackingCostDDP<ModelWrapperDDP<DYNAMICS_T>>(Q_, R_, numTimesteps_);
  terminal_cost_ = new TrackingTerminalCost<ModelWrapperDDP<DYNAMICS_T>>(Qf_);
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::computeFeedbackGains(Eigen::MatrixXf state)
{
  Eigen::MatrixXf control_traj = Eigen::MatrixXf::Zero(CONTROL_DIM, numTimesteps_);
  for (int t = 0; t < numTimesteps_; t++){
    for (int i = 0; i < CONTROL_DIM; i++){
      control_traj(i,t) = control_solution_[CONTROL_DIM*t + i];
    }
  }
  run_cost_->setTargets(state_solution_.data(), control_solution_.data(), numTimesteps_);
  terminal_cost_->xf = run_cost_->traj_target_x_.col(numTimesteps_ - 1);
  result_ = ddp_solver_->run(state, control_traj, *ddp_model_, *run_cost_, *terminal_cost_, U_MIN_, U_MAX_);
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
OptimizerResult<ModelWrapperDDP<DYNAMICS_T>> MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::getFeedbackGains()
{
  return result_;
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::resetControls()
{
  int i,j;
  //Set all the control values to their initial settings.
  for (i = 0; i < numTimesteps_; i++) {
    for (j = 0; j < CONTROL_DIM; j++) {
      U_[i*CONTROL_DIM + j] = init_u_[j];
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::cutThrottle()
{
  costs_->params_.desired_speed = 0.0;
  model_->control_rngs_[1].y = 0.0; //Max throttle to zero
  costs_->paramsToDevice();
  model_->paramsToDevice();
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::savitskyGolay()
{
  int i,j;
  Eigen::MatrixXf filter(1,5);
  Eigen::MatrixXf U_smoothed = Eigen::MatrixXf::Zero(8 + numTimesteps_, CONTROL_DIM);

  filter << -3, 12, 17, 12, -3;
  filter /= 35.0;
  for (i = 0; i < numTimesteps_ + 4; i++){
    if (i < 2) {
      for (j = 0; j < CONTROL_DIM; j++){
        U_smoothed(i, j) = control_hist_[CONTROL_DIM*i + j];
      }
    }
    else if (i < numTimesteps_ + 2) {
      for (j = 0; j < CONTROL_DIM; j++){
        U_smoothed(i,j) = U_[CONTROL_DIM*(i - 2) + j];
      }
    }
    else{
      for (j = 0; j < CONTROL_DIM; j++) {
        U_smoothed(i, j) = U_[CONTROL_DIM*(numTimesteps_ - 1) + j];
      }
    }
  }
  for (i = 0; i < numTimesteps_; i++){
    for (j = 0; j < CONTROL_DIM; j++){
      U_[CONTROL_DIM*i + j] = (filter*U_smoothed.block<5,1>(i,j))(0,0);
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::computeNominalTraj(Eigen::Matrix<float, STATE_DIM, 1> state)
{
  int i,j;
  Eigen::MatrixXf s(7,1);
  Eigen::MatrixXf u(2,1);
  s = state;
  for (i = 0; i < numTimesteps_; i++){
    for (j = 0; j < STATE_DIM; j++){
      //Set the current state solution
      state_solution_[i*STATE_DIM + j] = s(j);
    }
    u << U_[2*i], U_[2*i + 1];
    model_->updateState(s,u);
    //Set current control solution after clamping
    control_solution_[2*i] = u(0);
    control_solution_[2*i + 1] = u(1);
  }
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::slideControlSeq(int stride)
{
  //Slide the control sequence down by stride
  if (stride == 1){
    control_hist_[0] = control_hist_[2];
    control_hist_[1] = control_hist_[3];
    control_hist_[2] = U_[0];
    control_hist_[3] = U_[1];
  }
  else{
    int t = stride - 2;
    for (int i = 0; i < 4; i++){
      control_hist_[i] = U_[t + i];
    }
  }

  for (int i = 0; i < numTimesteps_- stride; i++) {
    for (int j = 0; j < CONTROL_DIM; j++) {
      U_[i*CONTROL_DIM + j] = U_[(i+stride)*CONTROL_DIM + j];
    }
  }
  //Initialize new controls to the init_u_ variable.
  for (int j = 1; j <= stride; j++) {
    for (int i = 0; i < CONTROL_DIM; i++){
      U_[(numTimesteps_ - j)*CONTROL_DIM + i] = init_u_[i];
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::computeControl(Eigen::Matrix<float, STATE_DIM, 1> state)
{
  //First transfer the state and current control sequence to the device.
  costs_->paramsToDevice();
  model_->paramsToDevice();

  HANDLE_ERROR( hipMemcpyAsync(state_d_, state.data(), STATE_DIM*sizeof(float), hipMemcpyHostToDevice, stream_));
  for (int opt_iter = 0; opt_iter < num_iters_; opt_iter++) {
    HANDLE_ERROR( hipMemcpyAsync(U_d_, U_.data(), CONTROL_DIM*numTimesteps_*sizeof(float), hipMemcpyHostToDevice, stream_));    
    //Generate a bunch of random numbers
    hiprandGenerateNormal(gen_, du_d_, NUM_ROLLOUTS*numTimesteps_*CONTROL_DIM, 0.0, 1.0);
    //Launch the rollout kernel
    launchRolloutKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>(numTimesteps_, state_d_, U_d_, du_d_, nu_d_, traj_costs_d_, model_, 
                        costs_, optimizationStride_, stream_);
    HANDLE_ERROR(hipMemcpyAsync(traj_costs_.data(), traj_costs_d_, NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost, stream_));
    //NOTE: The calls to hipMemcpyAsync are only asynchronous with regards to (1) CPU operations AND (2) GPU operations 
    //that are potentially occuring on other streams. Since all the previous kernel/memcpy operations use the same 
    //stream, they all occur sequentially with respect to our stream (which is necessary for correct execution)
  
    //Synchronize stream here since we want to do computations on the CPU
    HANDLE_ERROR( hipStreamSynchronize(stream_) );

    //Compute the baseline (minimum) sampled cost
    float baseline = traj_costs_[0];
    for (int i = 0; i < NUM_ROLLOUTS; i++) {
      if (traj_costs_[i] < baseline){
        baseline = traj_costs_[i];
      }
    }

    //Now resume GPU computations
    launchNormExpKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>(traj_costs_d_, gamma_, baseline, stream_);
    HANDLE_ERROR(hipMemcpyAsync(traj_costs_.data(), traj_costs_d_, NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost, stream_));
    hipStreamSynchronize(stream_);

    //Compute the normalizing term
    normalizer_ = 0;
    for (int i = 0; i < NUM_ROLLOUTS; i++) {
      normalizer_ += traj_costs_[i];
    }

    //Compute the cost weighted avergage.
    launchWeightedReductionKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>(traj_costs_d_, du_d_, nu_d_, normalizer_, numTimesteps_, stream_);

    //Transfer control update to host.
    HANDLE_ERROR( hipMemcpyAsync(du_.data(), du_d_, numTimesteps_*CONTROL_DIM*sizeof(float), hipMemcpyDeviceToHost, stream_));
    hipStreamSynchronize(stream_);

    //Save the control update
    for (int i = 0; i < numTimesteps_; i++) {
      for (int j = 0; j < CONTROL_DIM; j++) {
        U_[i*CONTROL_DIM + j] = du_[i*CONTROL_DIM + j];
      }
    }

  }
  //Smooth for the next optimization round
  savitskyGolay();
  //Compute the planned trajectory
  computeNominalTraj(state);
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
std::vector<float> MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::getControlSeq()
{
  return control_solution_;
}

template<class DYNAMICS_T, class COSTS_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
std::vector<float> MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::getStateSeq()
{
  return state_solution_;
}
