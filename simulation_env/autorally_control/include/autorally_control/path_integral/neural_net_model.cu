#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
* Copyright (c) 2013, Georgia Institute of Technology
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice, this
* list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright notice,
* this list of conditions and the following disclaimer in the documentation
* and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/**********************************************
* @file neural_net_model.cu
* @author Grady Williams <gradyrw@gmail.com>
* @date June 30, 2017
* @copyright 2016 Georgia Institute of Technology
* @brief NeuralNetModel class implementation.
*
***********************************************/

#define MPPI_NNET_NONLINEARITY(ans) tanh(ans)
#define MPPI_NNET_NONLINEARITY_DERIV(ans) (1 - powf(tanh(ans), 2))

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::NeuralNetModel(float delta_t, float2* control_rngs)
{
  dt_ = delta_t;
  if (control_rngs == NULL){
    control_rngs_ = new float2[CONTROL_DIM];
    for (int i = 0; i < CONTROL_DIM; i++){
      control_rngs_[i].x = -FLT_MAX;
      control_rngs_[i].y = FLT_MAX;
    }
  }
  else {
    control_rngs_ = control_rngs;
  }
  HANDLE_ERROR( hipMalloc((void**)&control_rngs_d_, CONTROL_DIM*sizeof(float2)) );

  //Initialize GPU memory for constraints and model parameters.
  HANDLE_ERROR( hipMalloc((void**)&theta_d_, NUM_PARAMS*sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void**)&stride_idcs_d_, (2*NUM_LAYERS + 1)*sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&net_structure_d_, NUM_LAYERS*sizeof(int)) );
  //Initialize the host matrices of weights and biases
  weights_ = new Eigen::Matrix<float, -1, -1, Eigen::RowMajor>[NUM_LAYERS-1];
  biases_ = new Eigen::Matrix<float, -1, -1, Eigen::RowMajor>[NUM_LAYERS-1];
  net_params_ = new float[NUM_PARAMS];

  weighted_in_ = new Eigen::MatrixXf[NUM_LAYERS - 1];
  for (int i = 1; i < NUM_LAYERS; i++){
    weighted_in_[i-1] = Eigen::MatrixXf::Zero(net_structure_[i], 1);
  }
}

template<int S_DIM, int C_DIM,  int K_DIM, int... layer_args>
NeuralNetModel<S_DIM, C_DIM,  K_DIM, layer_args...>::~NeuralNetModel()
{}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::loadParams(std::string model_path)
{
  int i,j,k;
  std::string bias_name = "";
  std::string weight_name = "";
  if (!fileExists(model_path)){
    ROS_FATAL("Could not load neural net model at path: %s", model_path.c_str());
  }
  cnpy::npz_t param_dict = cnpy::npz_load(model_path);
  for (i = 1; i < NUM_LAYERS; i++){
    bias_name = "dynamics_b" + std::to_string(i);
    weight_name = "dynamics_W" + std::to_string(i);
    cnpy::NpyArray weight_i_raw = param_dict[weight_name];
    cnpy::NpyArray bias_i_raw = param_dict[bias_name];
    double* weight_i = weight_i_raw.data<double>();
    double* bias_i = bias_i_raw.data<double>();
    //Copy the data into eigen arrays.
    Eigen::MatrixXf weight_i_mat(net_structure_[i], net_structure_[i-1]);
    Eigen::MatrixXf bias_i_vec(net_structure_[i], 1);
    for (j = 0; j < net_structure_[i]; j++){
      for (k = 0; k < net_structure_[i-1]; k++){
        weight_i_mat(j,k) = (float)weight_i[j*net_structure_[i-1] + k];
      }
    }
    for (j = 0; j < net_structure_[i]; j++){
      bias_i_vec(j,0) = (float)bias_i[j];
    }
    weights_[i-1] = weight_i_mat;
    biases_[i-1] = bias_i_vec;
  }
  //Save parameters to GPU memory
  paramsToDevice();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::setParams(Eigen::Matrix<float, -1, -1, Eigen::RowMajor>* weights, 
                                              Eigen::Matrix<float, -1, -1, Eigen::RowMajor>* biases)
{
  int i;
  for (i = 0; i < NUM_LAYERS - 1; i++){
    weights_[i] = weights[i];
    biases_[i] = biases[i];
  }
  paramsToDevice();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::paramsToDevice()
{
  int i,j,k;
  int stride = 0;
  for (i = 0; i < NUM_LAYERS - 1; i++){
    //stride index for the weight matrix
    stride_idcs_[2*i] = stride;
    for (j = 0; j < net_structure_[i+1]; j++){
      for (k = 0; k < net_structure_[i]; k++){
        net_params_[stride + j*net_structure_[i] + k] = weights_[i](j,k);
      }
    }
    stride += net_structure_[i+1]*net_structure_[i];
    //stride index for the bias
    stride_idcs_[2*i + 1] = stride;
    for (j = 0; j < net_structure_[i+1]; j++){
      net_params_[stride + j] = biases_[i](j,0);
    }
    stride += net_structure_[i+1];
  }
  stride_idcs_[NUM_LAYERS*2] = stride;
  HANDLE_ERROR( hipMemcpy(theta_d_, net_params_, NUM_PARAMS*sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(stride_idcs_d_, &stride_idcs_, (2*NUM_LAYERS + 1)*sizeof(int), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(net_structure_d_, &net_structure_, NUM_LAYERS*sizeof(int), hipMemcpyHostToDevice) );
//If we're using constant memory transfer theta_d_ to it.
#if defined(MPPI_NNET_USING_CONSTANT_MEM___)
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(NNET_PARAMS), theta_d_, NUM_PARAMS*sizeof(float)) );
#endif /*MPPI_NNET_USING_CONSTANT_MEM___*/
  HANDLE_ERROR( hipMemcpy(control_rngs_d_, control_rngs_, CONTROL_DIM*sizeof(float2), hipMemcpyHostToDevice) );
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::updateModel(std::vector<int> description, std::vector<float> data)
{
  //First make sure that the network structure matches the current network structure
  bool validUpdate = true;
  for (int i = 0; i < description.size(); i++){
    if (description[i] != net_structure_[i]){
      validUpdate = false;
    }
  }
  //If the network structure is valid save the parameters
  if (validUpdate){
    int stride = 0;
    for (int i = 0; i < NUM_LAYERS - 1; i++){
      for (int j = 0; j < net_structure_[i+1]; j++){
        for (int k = 0; k < net_structure_[i]; k++){
          weights_[i](j,k) = data[stride + j*net_structure_[i] + k];
        }
      }
      stride += net_structure_[i+1]*net_structure_[i];
    }
    for (int i = 0; i < NUM_LAYERS - 1; i++){
      for (int j = 0; j < net_structure_[i+1]; j++){
        biases_[i](j,0) = data[stride + j];
      }
      stride += net_structure_[i+1];
    }
  }
}

template<int S_DIM, int C_DIM,  int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM,  K_DIM, layer_args...>::printParamVec()
{
  int i;
  for (i = 0; i < NUM_PARAMS; i++){
    printf("Buffer Idx: %d, Value: %f \n", i, net_params_[i]);
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeKinematics(Eigen::MatrixXf &state)
{
  state_der_(0) = cosf(state(2))*state(4) - sinf(state(2))*state(5);
  state_der_(1) = sinf(state(2))*state(4) + cosf(state(2))*state(5);
  state_der_(2) = -state(6); //Pose estimate actually gives the negative yaw derivative
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeDynamics(Eigen::MatrixXf &state, Eigen::MatrixXf &control)
{
  int i,j;
  Eigen::MatrixXf acts(net_structure_[0], 1);
  for (i = 0; i < DYNAMICS_DIM; i++){
    acts(i) = state(i + (STATE_DIM - DYNAMICS_DIM));
  }
  for (i = 0; i < CONTROL_DIM; i++){
    acts(DYNAMICS_DIM + i) = control(i);
  }
  for (i = 0; i < NUM_LAYERS - 1; i++){
    weighted_in_[i] = (weights_[i]*acts + biases_[i]).eval();
    acts = Eigen::MatrixXf::Zero(net_structure_[i+1], 1);
    if (i < NUM_LAYERS - 2) { //Last layer doesn't apply any non-linearity
      for (j = 0; j < net_structure_[i+1]; j++){
        acts(j) = MPPI_NNET_NONLINEARITY( (weighted_in_[i])(j) ); //Nonlinear component.
      }
    }
    else {
      for (j = 0; j < net_structure_[i+1]; j++){
        acts(j) = (weighted_in_[i])(j) ;
      }
    }
  }
  for (i = 0; i < DYNAMICS_DIM; i++){
    state_der_(i + (STATE_DIM - DYNAMICS_DIM)) = acts(i);
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeGrad(Eigen::MatrixXf &state, Eigen::MatrixXf &control)
{
  jac_ = Eigen::Matrix<float, STATE_DIM, STATE_DIM + CONTROL_DIM>::Zero();

  //Start with the kinematic and physics model derivatives
  jac_.row(0) << 0, 0, -sin(state(2))*state(4) - cos(state(2))*state(5), 0, cos(state(2)), -sin(state(2)), 0, 0, 0;
  jac_.row(1) << 0, 0, cos(state(2))*state(4) - sin(state(2))*state(5), 0, sin(state(2)), cos(state(2)), 0, 0, 0;
  jac_.row(2) << 0, 0, 0, 0, 0, 0, -1, 0, 0;

  //First do the forward pass
  computeDynamics(state, control);

  //Start backprop
  ip_delta_ = Eigen::MatrixXf::Identity(DYNAMICS_DIM, DYNAMICS_DIM);
  Eigen::MatrixXf temp_delta = Eigen::MatrixXf::Identity(DYNAMICS_DIM, DYNAMICS_DIM);

  //Main backprop loop
  for (int i = NUM_LAYERS-2; i > 0; i--){
    Eigen::MatrixXf zp = weighted_in_[i-1];
    for (int j = 0; j < net_structure_[i]; j++){
      zp(j) = MPPI_NNET_NONLINEARITY_DERIV(zp(j));
    }
    ip_delta_ =  ( (weights_[i]).transpose()*ip_delta_ ).eval();
    for (int j = 0; j < DYNAMICS_DIM; j++){
      ip_delta_.col(j) = ip_delta_.col(j).array() * zp.array();
    }
  }
  //Finish the backprop loop
  ip_delta_ = ( ((weights_[0]).transpose())*ip_delta_).eval();
  jac_.bottomRightCorner(DYNAMICS_DIM, DYNAMICS_DIM + CONTROL_DIM) += ip_delta_.transpose();
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::enforceConstraints(Eigen::MatrixXf &state, Eigen::MatrixXf &control)
{
  int i;
  for (i = 0; i < CONTROL_DIM; i++){
    if (control(i) < control_rngs_[i].x){
      control(i) = control_rngs_[i].x;
    }
    else if (control(i) > control_rngs_[i].y){
      control(i) = control_rngs_[i].y;
    }
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::updateState(Eigen::MatrixXf &state, Eigen::MatrixXf &control)
{
  enforceConstraints(state, control);
  computeKinematics(state);
  computeDynamics(state, control);
  state += state_der_*dt_;
  state_der_ *= 0;
}

template<int S_DIM, int C_DIM,  int K_DIM, int... layer_args>
void NeuralNetModel<S_DIM, C_DIM,  K_DIM, layer_args...>::freeCudaMem()
{
  HANDLE_ERROR(hipFree(theta_d_));
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::cudaInit(float* theta_s)
{}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::printCudaParamVec()
{
  int i;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    for (i = 0; i < NUM_PARAMS; i++){
      printf("Buffer Idx: %d, Value: %f \n", i, theta_d_[i]);
    }
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::enforceConstraints(float* state, float* control)
{
  int i;
  for (i = 0; i < CONTROL_DIM; i++){
    if (control[i] < control_rngs_d_[i].x){
      control[i] = control_rngs_d_[i].x;
    }
    else if (control[i] > control_rngs_d_[i].y){
      control[i] = control_rngs_d_[i].y;
    }
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeStateDeriv(float* state, float* control, float* state_der, float* theta_s)
{
  if (threadIdx.y == 0){
    computeKinematics(state, state_der);
  }
  computeDynamics(state, control, state_der, theta_s);
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::incrementState(float* state, float* state_der)
{
  int i;
  int tdy = threadIdx.y;
  //Add the state derivative time dt to the current state.
  for (i = tdy; i < STATE_DIM; i+=blockDim.y){
    state[i] += state_der[i]*dt_;
    state_der[i] = 0; //Important: reset the state derivative to zero.
  }
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeKinematics(float* state, float* state_der)
{
  state_der[0] = cosf(state[2])*state[4] - sinf(state[2])*state[5];
  state_der[1] = sinf(state[2])*state[4] + cosf(state[2])*state[5];
  state_der[2] = -state[6]; //Pose estimate actually gives the negative yaw derivative
}

template<int S_DIM, int C_DIM, int K_DIM, int... layer_args>
__device__ void NeuralNetModel<S_DIM, C_DIM, K_DIM, layer_args...>::computeDynamics(float* state, float* control, float* state_der, float* theta_s)
{
  float* curr_act;
  float* next_act;
  float* tmp_act;
  float tmp;
  float* W;
  float* b;
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  int tdz = threadIdx.z;
  int i,j,k;
  curr_act = &theta_s[(2*LARGEST_LAYER)*(blockDim.x*tdz + tdx)];
  next_act = &theta_s[(2*LARGEST_LAYER)*(blockDim.x*tdz + tdx) + LARGEST_LAYER];
  for (i = tdy; i < DYNAMICS_DIM; i+= blockDim.y){
    curr_act[i] = state[i + (STATE_DIM - DYNAMICS_DIM)];
  }
  for (i = tdy; i < CONTROL_DIM; i+= blockDim.y){
    curr_act[DYNAMICS_DIM + i] = control[i];
  }
  __syncthreads();
  for (i = 0; i < NUM_LAYERS - 1; i++){
  //Conditional compilation depending on if we're using a global constant memory array or not.
  #if defined(MPPI_NNET_USING_CONSTANT_MEM___) //Use constant memory.
    W = &NNET_PARAMS[stride_idcs_d_[2*i]];
    b = &NNET_PARAMS[stride_idcs_d_[2*i + 1]];
  #else //Use (slow) global memory.
    W = &theta_d_[stride_idcs_d_[2*i]];
    b = &theta_d_[stride_idcs_d_[2*i + 1]];
  #endif
    for (j = tdy; j < net_structure_d_[i+1]; j += blockDim.y) {
      tmp = 0;
      for (k = 0; k < net_structure_d_[i]; k++){
        //No atomic add necessary.
        tmp += W[j*net_structure_d_[i] + k]*curr_act[k];
      }
      tmp += b[j];
      if (i < NUM_LAYERS - 2){
        tmp = MPPI_NNET_NONLINEARITY(tmp);
      }
      next_act[j] = tmp;
    }
    //Swap the two pointers
    tmp_act = curr_act;
    curr_act = next_act;
    next_act = tmp_act;
    __syncthreads();
  }
  for (i = tdy; i < DYNAMICS_DIM; i+= blockDim.y){
    state_der[i + (STATE_DIM - DYNAMICS_DIM)] = curr_act[i];
  }
  __syncthreads();
}
